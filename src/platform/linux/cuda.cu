#include "hip/hip_runtime.h"
// #include <algorithm>
#include <hip/hip_vector_types.h>
#include <limits>
#include <memory>
#include <optional>
#include <string_view>

#include "hip/hip_runtime.h"

using namespace std::literals;

#define SUNSHINE_STRINGVIEW_HELPER(x) x##sv
#define SUNSHINE_STRINGVIEW(x) SUNSHINE_STRINGVIEW_HELPER(x)

#define CU_CHECK(x, y) \
  if(check((x), SUNSHINE_STRINGVIEW(y ": "))) return -1

#define CU_CHECK_VOID(x, y) \
  if(check((x), SUNSHINE_STRINGVIEW(y ": "))) return;

#define CU_CHECK_PTR(x, y) \
  if(check((x), SUNSHINE_STRINGVIEW(y ": "))) return nullptr;

#define CU_CHECK_OPT(x, y) \
  if(check((x), SUNSHINE_STRINGVIEW(y ": "))) return std::nullopt;

#define CU_CHECK_IGNORE(x, y) \
  check((x), SUNSHINE_STRINGVIEW(y ": "))

using namespace std::literals;

//////////////////// Special desclarations
/**
 * NVCC segfaults when including <chrono>
 * Therefore, some declarations need to be added explicitely
 */
namespace platf {
struct img_t {
public:
  std::uint8_t *data {};
  std::int32_t width {};
  std::int32_t height {};
  std::int32_t pixel_pitch {};
  std::int32_t row_pitch {};

  virtual ~img_t() = default;
};
} // namespace platf

namespace video {
using __float4 = float[4];
using __float3 = float[3];
using __float2 = float[2];

struct __attribute__((__aligned__(16))) color_t {
  float4 color_vec_y;
  float4 color_vec_u;
  float4 color_vec_v;
  float2 range_y;
  float2 range_uv;
};

struct __attribute__((__aligned__(16))) color_extern_t {
  __float4 color_vec_y;
  __float4 color_vec_u;
  __float4 color_vec_v;
  __float2 range_y;
  __float2 range_uv;
};

static_assert(sizeof(video::color_t) == sizeof(video::color_extern_t), "color matrix struct mismatch");

extern color_t colors[4];
} // namespace video

//////////////////// End special declarations

namespace cuda {
auto constexpr INVALID_TEXTURE = std::numeric_limits<hipTextureObject_t>::max();

template<class T>
inline T div_align(T l, T r) {
  return (l + r - 1) / r;
}

void pass_error(const std::string_view &sv, const char *name, const char *description);
inline static int check(hipError_t result, const std::string_view &sv) {
  if(result) {
    auto name        = hipGetErrorName(result);
    auto description = hipGetErrorString(result);

    pass_error(sv, name, description);
    return -1;
  }

  return 0;
}

template<class T>
ptr_t make_ptr() {
  void *p;
  CU_CHECK_PTR(hipMalloc(&p, sizeof(T)), "Couldn't allocate color matrix");

  ptr_t ptr { p };

  return ptr;
}

void freeCudaPtr_t::operator()(void *ptr) {
  CU_CHECK_IGNORE(hipFree(ptr), "Couldn't free cuda device pointer");
}

void freeCudaStream_t::operator()(hipStream_t ptr) {
  CU_CHECK_IGNORE(hipStreamDestroy(ptr), "Couldn't free cuda stream");
}

stream_t make_stream(int flags) {
  hipStream_t stream;

  if(!flags) {
    CU_CHECK_PTR(hipStreamCreate(&stream), "Couldn't create cuda stream");
  }
  else {
    CU_CHECK_PTR(hipStreamCreateWithFlags(&stream, flags), "Couldn't create cuda stream with flags");
  }

  return stream_t { stream };
}

inline __device__ float3 bgra_to_rgb(uchar4 vec) {
  return make_float3((float)vec.z, (float)vec.y, (float)vec.x);
}

inline __device__ float3 bgra_to_rgb(float4 vec) {
  return make_float3(vec.z, vec.y, vec.x);
}

inline __device__ float2 calcUV(float3 pixel, const video::color_t *const color_matrix) {
  float4 vec_u = color_matrix->color_vec_u;
  float4 vec_v = color_matrix->color_vec_v;

  float u = dot(pixel, make_float3(vec_u)) + vec_u.w;
  float v = dot(pixel, make_float3(vec_v)) + vec_v.w;

  u = u * color_matrix->range_uv.x + color_matrix->range_uv.y;
  v = (v * color_matrix->range_uv.x + color_matrix->range_uv.y) * 224.0f / 256.0f + 0.0625f;

  return make_float2(u, v);
}

inline __device__ float calcY(float3 pixel, const video::color_t *const color_matrix) {
  float4 vec_y = color_matrix->color_vec_y;

  return (dot(pixel, make_float3(vec_y)) + vec_y.w) * color_matrix->range_y.x + color_matrix->range_y.y;
}

__global__ void RGBA_to_NV12(
  hipTextureObject_t srcImage, std::uint8_t *dstY, std::uint8_t *dstUV,
  std::uint32_t dstPitchY, std::uint32_t dstPitchUV,
  float scale, const viewport_t viewport, const video::color_t *const color_matrix) {

  int idX = (threadIdx.x + blockDim.x * blockIdx.x) * 2;
  int idY = (threadIdx.y + blockDim.y * blockIdx.y);

  if(idX >= viewport.width) return;
  if(idY >= viewport.height) return;

  float x = idX * scale;
  float y = idY * scale;

  idX += viewport.offsetX;
  idY += viewport.offsetY;

  dstY  = dstY + idX + idY * dstPitchY;
  dstUV = dstUV + idX + (idY / 2 * dstPitchUV);

  float3 rgb_l = bgra_to_rgb(tex2D<float4>(srcImage, x, y));
  float3 rgb_r = bgra_to_rgb(tex2D<float4>(srcImage, x + scale, y));

  float2 uv = calcUV((rgb_l + rgb_r) * 0.5f, color_matrix) * 256.0f;

  dstUV[0] = uv.x;
  dstUV[1] = uv.y;
  dstY[0]  = calcY(rgb_l, color_matrix) * 245.0f; // 245.0f is a magic number to ensure slight changes in luminosity are more visisble
  dstY[1]  = calcY(rgb_r, color_matrix) * 245.0f; // 245.0f is a magic number to ensure slight changes in luminosity are more visisble
}

int tex_t::copy(std::uint8_t *src, int height, int pitch) {
  CU_CHECK(hipMemcpy2DToArray(array, 0, 0, src, pitch, pitch, height, hipMemcpyDeviceToDevice), "Couldn't copy to cuda array from deviceptr");

  return 0;
}

std::optional<tex_t> tex_t::make(int height, int pitch) {
  tex_t tex;

  auto format = hipCreateChannelDesc<uchar4>();
  CU_CHECK_OPT(hipMallocArray(&tex.array, &format, pitch, height, hipArrayDefault), "Couldn't allocate cuda array");

  hipResourceDesc res {};
  res.resType         = hipResourceTypeArray;
  res.res.array.array = tex.array;

  hipTextureDesc desc {};

  desc.readMode         = hipReadModeNormalizedFloat;
  desc.filterMode       = hipFilterModePoint;
  desc.normalizedCoords = false;

  std::fill_n(std::begin(desc.addressMode), 2, hipAddressModeClamp);

  CU_CHECK_OPT(hipCreateTextureObject(&tex.texture.point, &res, &desc, nullptr), "Couldn't create cuda texture that uses point interpolation");

  desc.filterMode = hipFilterModeLinear;

  CU_CHECK_OPT(hipCreateTextureObject(&tex.texture.linear, &res, &desc, nullptr), "Couldn't create cuda texture that uses linear interpolation");

  return std::move(tex);
}

tex_t::tex_t() : array {}, texture { INVALID_TEXTURE } {}
tex_t::tex_t(tex_t &&other) : array { other.array }, texture { other.texture } {
  other.array          = 0;
  other.texture.point  = INVALID_TEXTURE;
  other.texture.linear = INVALID_TEXTURE;
}

tex_t &tex_t::operator=(tex_t &&other) {
  std::swap(array, other.array);
  std::swap(texture, other.texture);

  return *this;
}

tex_t::~tex_t() {
  if(texture.point != INVALID_TEXTURE) {
    CU_CHECK_IGNORE(hipDestroyTextureObject(texture.point), "Couldn't deallocate cuda texture that uses point interpolation");

    texture.point = INVALID_TEXTURE;
  }

  if(texture.linear != INVALID_TEXTURE) {
    CU_CHECK_IGNORE(hipDestroyTextureObject(texture.linear), "Couldn't deallocate cuda texture that uses linear interpolation");

    texture.linear = INVALID_TEXTURE;
  }

  if(array) {
    CU_CHECK_IGNORE(hipFreeArray(array), "Couldn't deallocate cuda array");

    array = hipArray_t {};
  }
}

sws_t::sws_t(int in_width, int in_height, int out_width, int out_height, int pitch, int threadsPerBlock, ptr_t &&color_matrix)
    : threadsPerBlock { threadsPerBlock }, color_matrix { std::move(color_matrix) } {
  // Ensure aspect ratio is maintained
  auto scalar       = std::fminf(out_width / (float)in_width, out_height / (float)in_height);
  auto out_width_f  = in_width * scalar;
  auto out_height_f = in_height * scalar;

  // result is always positive
  auto offsetX_f = (out_width - out_width_f) / 2;
  auto offsetY_f = (out_height - out_height_f) / 2;

  viewport.width  = out_width_f;
  viewport.height = out_height_f;

  viewport.offsetX = offsetX_f;
  viewport.offsetY = offsetY_f;

  scale = 1.0f / scalar;
}

std::optional<sws_t> sws_t::make(int in_width, int in_height, int out_width, int out_height, int pitch) {
  hipDeviceProp_t props;
  int device;
  CU_CHECK_OPT(hipGetDevice(&device), "Couldn't get cuda device");
  CU_CHECK_OPT(hipGetDeviceProperties(&props, device), "Couldn't get cuda device properties");

  auto ptr = make_ptr<video::color_t>();
  if(!ptr) {
    return std::nullopt;
  }

  return std::make_optional<sws_t>(in_width, in_height, out_width, out_height, pitch, props.maxThreadsPerMultiProcessor / props.maxBlocksPerMultiProcessor, std::move(ptr));
}

int sws_t::convert(std::uint8_t *Y, std::uint8_t *UV, std::uint32_t pitchY, std::uint32_t pitchUV, hipTextureObject_t texture, stream_t::pointer stream) {
  return convert(Y, UV, pitchY, pitchUV, texture, stream, viewport);
}

int sws_t::convert(std::uint8_t *Y, std::uint8_t *UV, std::uint32_t pitchY, std::uint32_t pitchUV, hipTextureObject_t texture, stream_t::pointer stream, const viewport_t &viewport) {
  int threadsX = viewport.width / 2;
  int threadsY = viewport.height;

  dim3 block(threadsPerBlock);
  dim3 grid(div_align(threadsX, threadsPerBlock), threadsY);

  RGBA_to_NV12<<<grid, block, 0, stream>>>(texture, Y, UV, pitchY, pitchUV, scale, viewport, (video::color_t *)color_matrix.get());

  return CU_CHECK_IGNORE(hipGetLastError(), "RGBA_to_NV12 failed");
}

void sws_t::set_colorspace(std::uint32_t colorspace, std::uint32_t color_range) {
  video::color_t *color_p;
  switch(colorspace) {
  case 5: // SWS_CS_SMPTE170M
    color_p = &video::colors[0];
    break;
  case 1: // SWS_CS_ITU709
    color_p = &video::colors[2];
    break;
  case 9: // SWS_CS_BT2020
  default:
    color_p = &video::colors[0];
  };

  if(color_range > 1) {
    // Full range
    ++color_p;
  }

  CU_CHECK_IGNORE(hipMemcpy(color_matrix.get(), color_p, sizeof(video::color_t), hipMemcpyHostToDevice), "Couldn't copy color matrix to cuda");
}

int sws_t::load_ram(platf::img_t &img, hipArray_t array) {
  return CU_CHECK_IGNORE(hipMemcpy2DToArray(array, 0, 0, img.data, img.row_pitch, img.width * img.pixel_pitch, img.height, hipMemcpyHostToDevice), "Couldn't copy to cuda array");
}

} // namespace cuda